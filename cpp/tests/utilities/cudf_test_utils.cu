/*
 * Copyright 2018 BlazingDB, Inc.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "cudf_test_utils.cuh"

void print_gdf_column(gdf_column const * the_column)
{
  const size_t num_rows = the_column->size;
  const gdf_dtype gdf_col_type = the_column->dtype;
  switch(gdf_col_type)
  {
    case GDF_INT8:
      {
        using col_type = int8_t;
        col_type * col_data = static_cast<col_type*>(the_column->data);
        print_typed_column<col_type>(col_data, the_column->valid, num_rows);
        break;
      }
    case GDF_INT16:
      {
        using col_type = int16_t;
        col_type * col_data = static_cast<col_type*>(the_column->data);
        print_typed_column<col_type>(col_data, the_column->valid, num_rows);
        break;
      }
    case GDF_INT32:
      {
        using col_type = int32_t;
        col_type * col_data = static_cast<col_type*>(the_column->data);
        print_typed_column<col_type>(col_data, the_column->valid, num_rows);
        break;
      }
    case GDF_INT64:
      {
        using col_type = int64_t;
        col_type * col_data = static_cast<col_type*>(the_column->data);
        print_typed_column<col_type>(col_data, the_column->valid, num_rows);
        break;
      }
    case GDF_FLOAT32:
      {
        using col_type = float;
        col_type * col_data = static_cast<col_type*>(the_column->data);
        print_typed_column<col_type>(col_data, the_column->valid, num_rows);
        break;
      }
    case GDF_FLOAT64:
      {
        using col_type = double;
        col_type * col_data = static_cast<col_type*>(the_column->data);
        print_typed_column<col_type>(col_data, the_column->valid, num_rows);
        break;
      }
    default:
      {
        std::cout << "Attempted to print unsupported type.\n";
      }
  }
}

void print_valid_data(const gdf_valid_type *validity_mask,
                      const size_t num_rows)
{
  hipError_t error;
  hipPointerAttribute_t attrib;
  hipPointerGetAttributes(&attrib, validity_mask);
  error = hipGetLastError();

  const size_t num_masks = gdf_get_num_chars_bitmask(num_rows);
  std::vector<gdf_valid_type> h_mask(num_masks);
  if (error != hipErrorInvalidValue && attrib.memoryType == hipMemoryTypeDevice)
    hipMemcpy(h_mask.data(), validity_mask, num_masks * sizeof(gdf_valid_type), hipMemcpyDeviceToHost);
  else
    memcpy(h_mask.data(), validity_mask, num_masks * sizeof(gdf_valid_type));

  std::transform(h_mask.begin(), h_mask.end(), std::ostream_iterator<std::string>(std::cout, " "), 
                 [](gdf_valid_type x){ 
                   auto bits = std::bitset<GDF_VALID_BITSIZE>(x).to_string('@'); 
                   return std::string(bits.rbegin(), bits.rend());  
                 });
  std::cout << std::endl;
}

